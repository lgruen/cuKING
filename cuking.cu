#include "hip/hip_runtime.h"
#include <absl/container/flat_hash_map.h>
#include <absl/flags/flag.h>
#include <absl/flags/parse.h>
#include <absl/status/status.h>
#include <absl/status/statusor.h>
#include <absl/strings/str_cat.h>
#include <absl/time/time.h>
#include <arrow/filesystem/filesystem.h>
#include <arrow/result.h>
#include <arrow/status.h>

#include <algorithm>
#include <iostream>
#include <memory>
#include <nlohmann/json.hpp>
#include <string>
#include <vector>

#include "utils.h"

ABSL_FLAG(std::string, input_uri, "",
          "URI containing the sample bit sets. Supports file:// as well as "
          "gs://, e.g. gs://some/bucket/my_table.parquet");
ABSL_FLAG(std::string, output_uri, "",
          "The sparse relatedness matrix JSON output path. Supports file:// as "
          "well as gs://, e.g. gs://some/bucket/relatedness.json");
ABSL_FLAG(
    uint32_t, max_results, 100 << 20,
    "How many coefficients for related sample pairs to reserve memory for.");
ABSL_FLAG(
    float, king_coeff_threshold, 0.0442f,
    "Only store coefficients larger than this threshold. Defaults to 3rd "
    "degree or closer (see https://www.kingrelatedness.com/manual.shtml).");

namespace cuking {

absl::Status ToAbslStatus(const arrow::Status &status) {
  return absl::UnknownError(status.ToString());
}

}  // namespace cuking

namespace {

// Custom deleter for RAII-style CUDA-managed array.
template <typename T>
struct CudaArrayDeleter {
  void operator()(T *const val) const { hipFree(val); }
};

template <typename T>
using CudaArray = std::unique_ptr<T[], CudaArrayDeleter<T>>;

template <typename T>
CudaArray<T> NewCudaArray(const size_t size) {
  static_assert(std::is_pod<T>::value, "A must be a POD type.");
  T *buffer = nullptr;
  const auto err = hipMallocManaged(&buffer, size * sizeof(T));
  if (err) {
    std::cerr << "Error: can't allocate CUDA memory: "
              << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  return CudaArray<T>(buffer, CudaArrayDeleter<T>());
}

__device__ float ComputeKing(const uint64_t *const het_i_entries,
                             const uint64_t *const hom_alt_i_entries,
                             const uint64_t *const het_j_entries,
                             const uint64_t *const hom_alt_j_entries,
                             const uint32_t num_entries) {
  // See https://hail.is/docs/0.2/methods/relatedness.html#hail.methods.king.
  uint32_t num_het_i = 0, num_het_j = 0, num_both_het = 0, num_opposing_hom = 0;
  for (uint32_t k = 0; k < num_entries; ++k) {
    const uint64_t het_i = het_i_entries[k];
    const uint64_t hom_alt_i = hom_alt_i_entries[k];
    const uint64_t het_j = het_j_entries[k];
    const uint64_t hom_alt_j = hom_alt_j_entries[k];
    const uint64_t hom_ref_i = (~het_i) & (~hom_alt_i);
    const uint64_t hom_ref_j = (~het_j) & (~hom_alt_j);
    const uint64_t missing_mask_i = ~(het_i & hom_alt_i);
    const uint64_t missing_mask_j = ~(het_j & hom_alt_j);
    const uint64_t missing_mask = missing_mask_i & missing_mask_j;
    num_het_i += __popcll(het_i & missing_mask_i);
    num_het_j += __popcll(het_j & missing_mask_j);
    num_both_het += __popcll(het_i & het_j & missing_mask);
    num_opposing_hom += __popcll(
        ((hom_ref_i & hom_alt_j) | (hom_ref_j & hom_alt_i)) & missing_mask);
  }

  // Return the "between-family" estimator.
  const uint32_t min_hets = num_het_i < num_het_j ? num_het_i : num_het_j;
  return 0.5f +
         (2.f * num_both_het - 4.f * num_opposing_hom - num_het_i - num_het_j) /
             (4.f * min_hets);
}

// Stores the KING coefficient for one pair of samples.
struct KingResult {
  uint32_t sample_i, sample_j;
  float coeff;
};

__global__ void ComputeKingKernel(const uint32_t num_samples,
                                  const uint32_t words_per_sample,
                                  const uint64_t *const bit_sets,
                                  const float coeff_threshold,
                                  const uint32_t max_results,
                                  KingResult *const results,
                                  uint32_t *const result_index) {
  const uint64_t index = uint64_t(blockIdx.x) * blockDim.x + threadIdx.x;
  const uint64_t i = index / num_samples;
  const uint64_t j = index % num_samples;
  if (i >= num_samples || i >= j) {
    return;
  }

  const uint32_t num_entries = words_per_sample / 2;
  const uint64_t offset_i = i * words_per_sample;
  const uint64_t offset_j = j * words_per_sample;
  const float coeff = ComputeKing(
      bit_sets + offset_i, bit_sets + offset_i + num_entries,
      bit_sets + offset_j, bit_sets + offset_j + num_entries, num_entries);

  if (coeff > coeff_threshold) {
    // Reserve a result slot atomically to avoid collisions.
    const uint32_t reserved = atomicAdd(result_index, 1u);
    if (reserved < max_results) {
      KingResult &result = results[reserved];
      result.sample_i = i;
      result.sample_j = j;
      result.coeff = coeff;
    }
  }
}

absl::Status Run() {
  // Validate flags.
  const auto input_uri = absl::GetFlag(FLAGS_input_uri);
  if (input_uri.empty()) {
    return absl::InvalidArgumentError("No input URI specified");
  }
  const auto output_uri = absl::GetFlag(FLAGS_output_uri);
  if (output_uri.empty()) {
    return absl::InvalidArgumentError("No output URI specified");
  }

  std::cout << "Reading metadata...";
  std::cout.flush();
  cuking::StopWatch stop_watch;
  std::string input_path;
  ASSIGN_OR_RETURN(const auto input_fs,
                   arrow::fs::FileSystemFromUri(input_uri, &input_path));
  ASSIGN_OR_RETURN(auto metadata_file, input_fs->OpenInputFile(absl::StrCat(
                                           input_path, "/metadata.json")));
  ASSIGN_OR_RETURN(const uint64_t metadata_file_size, metadata_file->GetSize());
  std::vector<uint8_t> metadata_buffer(metadata_file_size);
  ASSIGN_OR_RETURN(
      const uint64_t metadata_bytes_read,
      metadata_file->ReadAt(0, metadata_file_size, metadata_buffer.data()));
  if (metadata_bytes_read != metadata_file_size) {
    return absl::FailedPreconditionError(absl::StrCat(
        "Expected to read ", metadata_file_size, " metadata bytes, but read ",
        metadata_bytes_read, " bytes instead"));
  }
  const auto metadata = nlohmann::json::parse(
      metadata_buffer.begin(), metadata_buffer.end(),
      /* parser_callback_t */ nullptr, /* allow_exceptions */ false);
  if (metadata.is_discarded()) {
    return absl::FailedPreconditionError("Failed to parse metadata JSON");
  }
  std::cout << " (" << stop_watch.GetElapsedAndReset() << ")" << std::endl;

  const std::vector<std::string_view> sample_ids = metadata["samples"];
  const uint32_t num_samples = sample_ids.size();
  std::cout << "Metadata contains " << num_samples << " samples with "
            << metadata["num_sites"] << " sites." << std::endl;

  std::cout << "Reading bit sets...";
  std::cout.flush();
  ASSIGN_OR_RETURN(auto bit_set_file, input_fs->OpenInputFile(absl::StrCat(
                                          input_path, "/bit_set.bin")));
  ASSIGN_OR_RETURN(const uint64_t bit_set_file_size, bit_set_file->GetSize());
  const uint32_t words_per_sample = metadata["words_per_sample"];
  if (bit_set_file_size !=
      uint64_t(num_samples) * words_per_sample * sizeof(uint64_t)) {
    return absl::FailedPreconditionError(
        absl::StrCat("Unexpected bit set file size: ", bit_set_file_size));
  }
  auto bit_set =
      NewCudaArray<uint64_t>(uint64_t(num_samples) * words_per_sample);
  ASSIGN_OR_RETURN(const uint64_t bit_set_bytes_read,
                   bit_set_file->ReadAt(0, bit_set_file_size, bit_set.get()));
  if (bit_set_bytes_read != bit_set_file_size) {
    return absl::FailedPreconditionError(absl::StrCat(
        "Expected to read ", bit_set_file_size, " bit set bytes, but read ",
        bit_set_bytes_read, " bytes instead"));
  }
  std::cout << " (" << stop_watch.GetElapsedAndReset() << ")" << std::endl;

  std::cout << "Allocating memory for results...";
  std::cout.flush();
  const uint32_t kMaxResults = absl::GetFlag(FLAGS_max_results);
  auto results = NewCudaArray<KingResult>(kMaxResults);
  memset(results.get(), 0, sizeof(KingResult) * kMaxResults);
  // We just need a single value.
  auto result_index = NewCudaArray<uint32_t>(1);
  result_index[0] = 0;
  std::cout << " (" << stop_watch.GetElapsedAndReset() << ")" << std::endl;

  std::cout << "Running KING CUDA kernel...";
  std::cout.flush();
  constexpr uint64_t kCudaBlockSize = 1024;
  const uint64_t kNumCudaBlocks =
      cuking::CeilIntDiv(uint64_t(num_samples) * num_samples, kCudaBlockSize);
  ComputeKingKernel<<<kNumCudaBlocks, kCudaBlockSize>>>(
      num_samples, words_per_sample, bit_set.get(),
      absl::GetFlag(FLAGS_king_coeff_threshold), kMaxResults, results.get(),
      result_index.get());

  // Wait for GPU to finish before accessing on host.
  hipDeviceSynchronize();
  std::cout << " (" << stop_watch.GetElapsedAndReset() << ")" << std::endl;

  // Free some memory for postprocessing.
  bit_set.reset();

  const uint32_t num_results = result_index[0];
  if (num_results > kMaxResults) {
    return absl::ResourceExhaustedError(
        "Could not store all results: try increasing the --max_results "
        "parameter.");
  }

  std::cout << "Found " << num_results
            << " coefficients above the cut-off threshold." << std::endl;

  std::cout << "Processing results...";
  std::cout.flush();
  std::vector<bool> related(num_samples);
  for (uint32_t i = 0; i < num_results; ++i) {
    const auto &result = results[i];
    related[result.sample_i] = related[result.sample_j] = true;
  }

  uint32_t num_related = 0;
  for (size_t i = 0; i < num_samples; ++i) {
    if (related[i]) {
      ++num_related;
    }
  }

  std::cout << " (" << stop_watch.GetElapsedAndReset() << ")" << std::endl;
  std::cout << num_related << " related samples found." << std::endl;

  std::cout << "Writing output...";
  std::cout.flush();

  // Create a map for JSON serialization.
  absl::flat_hash_map<std::string_view,
                      absl::flat_hash_map<std::string_view, float>>
      result_map;
  for (size_t i = 0; i < num_results; ++i) {
    const auto &result = results[i];
    result_map[sample_ids[result.sample_i]][sample_ids[result.sample_j]] =
        result.coeff;
  }

  std::string output_path;
  ASSIGN_OR_RETURN(const auto output_fs,
                   arrow::fs::FileSystemFromUri(output_uri, &output_path));
  ASSIGN_OR_RETURN(auto output_stream,
                   output_fs->OpenOutputStream(output_path));
  RETURN_IF_ERROR(output_stream->Write(nlohmann::json(result_map).dump(4)));
  RETURN_IF_ERROR(output_stream->Close());
  std::cout << " (" << stop_watch.GetElapsedAndReset() << ")" << std::endl;

  return absl::OkStatus();
}

}  // namespace

int main(int argc, char **argv) {
  absl::ParseCommandLine(argc, argv);

  if (const auto status = Run(); !status.ok()) {
    std::cerr << std::endl << "Error: " << status << std::endl;
    return 1;
  }

  return 0;
}
