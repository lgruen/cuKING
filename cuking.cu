#include "hip/hip_runtime.h"
#include <absl/base/thread_annotations.h>
#include <absl/flags/flag.h>
#include <absl/flags/parse.h>
#include <absl/synchronization/blocking_counter.h>
#include <absl/synchronization/mutex.h>
#include <absl/time/time.h>
#include <absl/types/span.h>
#include <zstd.h>

#include <filesystem>
#include <fstream>
#include <iostream>
#include <memory>
#include <queue>
#include <string>
#include <thread>
#include <vector>

#include "cuking.h"

ABSL_FLAG(std::string, sample_list, "",
          "A text file listing one .cuking.zst input file path per line.");
ABSL_FLAG(
    size_t, sample_range_begin, 0,
    "The inclusive index of the first sample to consider in the sample list.");
ABSL_FLAG(
    size_t, sample_range_end, 0,
    "The exclusive index of the last sample to consider in the sample list.");
ABSL_FLAG(int, num_reader_threads, 100,
          "How many threads to use for parallel file reading.");
ABSL_FLAG(bool, run_on_cpu, false,
          "Whether to run computation on CPU, useful for validation.");

namespace {
// Adapted from the Abseil thread pool.
class ThreadPool {
 public:
  explicit ThreadPool(const int num_threads) {
    assert(num_threads > 0);
    for (int i = 0; i < num_threads; ++i) {
      threads_.push_back(std::thread(&ThreadPool::WorkLoop, this));
    }
  }

  ThreadPool(const ThreadPool &) = delete;
  ThreadPool &operator=(const ThreadPool &) = delete;

  ~ThreadPool() {
    {
      absl::MutexLock l(&mu_);
      for (size_t i = 0; i < threads_.size(); i++) {
        queue_.push(nullptr);  // Shutdown signal.
      }
    }
    for (auto &t : threads_) {
      t.join();
    }
  }

  // Schedule a function to be run on a ThreadPool thread immediately.
  void Schedule(std::function<void()> func) {
    assert(func != nullptr);
    absl::MutexLock l(&mu_);
    queue_.push(std::move(func));
  }

 private:
  bool WorkAvailable() const ABSL_EXCLUSIVE_LOCKS_REQUIRED(mu_) {
    return !queue_.empty();
  }

  void WorkLoop() {
    while (true) {
      std::function<void()> func;
      {
        absl::MutexLock l(&mu_);
        mu_.Await(absl::Condition(this, &ThreadPool::WorkAvailable));
        func = std::move(queue_.front());
        queue_.pop();
      }
      if (func == nullptr) {  // Shutdown signal.
        break;
      }
      func();
    }
  }

  absl::Mutex mu_;
  std::queue<std::function<void()>> queue_ ABSL_GUARDED_BY(mu_);
  std::vector<std::thread> threads_;
};

// Custom deleter for RAII-style CUDA-managed array.
template <typename T>
struct CudaArrayDeleter {
  void operator()(T *const val) const { hipFree(val); }
};

template <typename T>
using CudaArray = std::unique_ptr<T[], CudaArrayDeleter<T>>;

template <typename T>
CudaArray<T> NewCudaArray(const size_t size) {
  static_assert(std::is_pod<T>::value, "A must be a POD type.");
  T *buffer = nullptr;
  const auto err = hipMallocManaged(&buffer, size * sizeof(T));
  if (err) {
    std::cerr << "Error: can't allocate CUDA memory: "
              << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  return CudaArray<T>(buffer, CudaArrayDeleter<T>());
}

struct Sample {
  const uint16_t *entries;  // Not owned.
  uint32_t num_entries;
  uint32_t num_hets;
};

struct ReadSamplesResult {
  CudaArray<Sample> samples;
  std::vector<CudaArray<uint16_t>> buffers;
};

// Reads and decompresses sample data from `paths`, adding entries with
// corresponding index into `result`.  Returns false if any failures occurred.
bool ReadSamples(const absl::Span<std::string> &paths,
                 ThreadPool *const thread_pool,
                 ReadSamplesResult *const result) {
  result->samples = NewCudaArray<Sample>(paths.size());
  result->buffers.clear();
  result->buffers.resize(paths.size());

  absl::BlockingCounter blocking_counter(paths.size());
  std::atomic<bool> success(true);
  for (size_t i = 0; i < paths.size(); ++i) {
    thread_pool->Schedule([&, i] {
      const auto &path = paths[i];
      // Determine the file size.
      std::error_code error_code;
      const size_t file_size = std::filesystem::file_size(path, error_code);
      if (error_code) {
        std::cerr << "Error: failed to determine size of \"" << path
                  << "\": " << error_code << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      std::ifstream in(path, std::ifstream::binary);
      if (!in) {
        std::cerr << "Error: failed to open \"" << path << "\"." << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      // Read the entire file.
      std::vector<uint8_t> contents(file_size);
      in.read(reinterpret_cast<char *>(contents.data()), file_size);
      if (!in) {
        std::cerr << "Error: failed to read \"" << path << "\"." << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }
      in.close();

      // Validate the file header.
      cuking::FileHeader file_header;
      bool valid_header = true;
      if (contents.size() < sizeof(cuking::FileHeader)) {
        valid_header = false;
      } else {
        memcpy(&file_header, contents.data(), sizeof(cuking::FileHeader));
        for (size_t i = 0; i < sizeof(file_header.magic); ++i) {
          if (file_header.magic[i] != cuking::kExpectedMagic[i]) {
            valid_header = false;
            break;
          }
        }
      }

      if (!valid_header) {
        std::cerr << "Error: failed to validate header for \"" << path << "\"."
                  << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      // Decompress the contents.
      const size_t num_entries =
          file_header.decompressed_size / sizeof(uint16_t);
      result->buffers[i] = NewCudaArray<uint16_t>(num_entries);
      auto &sample = result->samples[i];
      sample.entries = result->buffers[i].get();
      sample.num_entries = num_entries;
      sample.num_hets = file_header.num_hets;
      const size_t zstd_result = ZSTD_decompress(
          result->buffers[i].get(), file_header.decompressed_size,
          contents.data() + sizeof(cuking::FileHeader),
          contents.size() - sizeof(cuking::FileHeader));
      if (ZSTD_isError(zstd_result) ||
          zstd_result != file_header.decompressed_size) {
        std::cerr << "Error: failed to decompress \"" << path
                  << "\": " << ZSTD_getErrorName(zstd_result) << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      blocking_counter.DecrementCount();
    });
  }

  blocking_counter.Wait();
  return success;
}

__device__ __host__ inline uint32_t DecodeLocusIndexDelta(
    const uint16_t value) {
  return value >> 1;
}

__device__ __host__ inline cuking::VariantCategory DecodeVariantCategory(
    const uint16_t value) {
  return static_cast<cuking::VariantCategory>(value & 1);
}

__device__ __host__ float ComputeKing(const Sample &sample_i,
                                      const Sample &sample_j) {
  // See https://hail.is/docs/0.2/methods/relatedness.html#hail.methods.king.
  uint32_t num_both_het = 0, num_opposing_hom = 0;
  for (uint32_t index_i = 0, index_j = 0,
                pos_i = DecodeLocusIndexDelta(sample_i.entries[0]),
                pos_j = DecodeLocusIndexDelta(sample_j.entries[0]);
       pos_i != static_cast<uint32_t>(-1) ||
       pos_j != static_cast<uint32_t>(-1);) {
    if (pos_i < pos_j) {
      if (DecodeVariantCategory(sample_i.entries[index_i]) ==
          cuking::VariantCategory::kHomAlt) {
        ++num_opposing_hom;
      }

      if (++index_i < sample_i.num_entries) {
        pos_i += DecodeLocusIndexDelta(sample_i.entries[index_i]);
      } else {
        pos_i = static_cast<uint32_t>(-1);
      }
    }

    if (pos_j < pos_i) {
      if (DecodeVariantCategory(sample_j.entries[index_j]) ==
          cuking::VariantCategory::kHomAlt) {
        ++num_opposing_hom;
      }

      if (++index_j < sample_j.num_entries) {
        pos_j += DecodeLocusIndexDelta(sample_j.entries[index_j]);
      } else {
        pos_j = static_cast<uint32_t>(-1);
      }
    }

    if (pos_i == pos_j && pos_i != static_cast<uint32_t>(-1)) {
      if (DecodeVariantCategory(sample_i.entries[index_i]) ==
              cuking::VariantCategory::kHet &&
          DecodeVariantCategory(sample_j.entries[index_j]) ==
              cuking::VariantCategory::kHet) {
        ++num_both_het;
      }

      if (++index_i < sample_i.num_entries) {
        pos_i += DecodeLocusIndexDelta(sample_i.entries[index_i]);
      } else {
        pos_i = static_cast<uint32_t>(-1);
      }

      if (++index_j < sample_j.num_entries) {
        pos_j += DecodeLocusIndexDelta(sample_j.entries[index_j]);
      } else {
        pos_j = static_cast<uint32_t>(-1);
      }
    }
  }

  // Return the "between-family" estimator.
  const uint32_t min_hets = sample_i.num_hets < sample_j.num_hets
                                ? sample_i.num_hets
                                : sample_j.num_hets;
  return 0.5f + (2.f * num_both_het - 4.f * num_opposing_hom -
                 sample_i.num_hets - sample_j.num_hets) /
                    (4.f * min_hets);
}

__global__ void ComputeKingKernel(const Sample *const samples,
                                  const int num_samples, float *const result) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int i = index / num_samples;
  const int j = index % num_samples;
  if (i >= num_samples || i >= j) {
    return;
  }
  result[i * num_samples + j] = ComputeKing(samples[i], samples[j]);
}

}  // namespace

int main(int argc, char **argv) {
  absl::ParseCommandLine(argc, argv);

  const auto &sample_list_file = absl::GetFlag(FLAGS_sample_list);
  if (sample_list_file.empty()) {
    std::cerr << "Error: no sample list file specified." << std::endl;
    return 1;
  }

  std::ifstream sample_list(sample_list_file);
  std::string line;
  std::vector<std::string> sample_paths;
  while (std::getline(sample_list, line)) {
    if (line.empty()) {
      continue;
    }
    sample_paths.push_back(line);
  }

  const size_t sample_range_begin = absl::GetFlag(FLAGS_sample_range_begin);
  const size_t sample_range_end = absl::GetFlag(FLAGS_sample_range_end);
  if (sample_range_begin >= sample_range_end ||
      sample_range_end > sample_paths.size()) {
    std::cerr << "Error: invalid sample range specified." << std::endl;
    return 1;
  }

  const size_t num_samples = sample_range_end - sample_range_begin;
  const auto sample_paths_span =
      absl::MakeSpan(sample_paths).subspan(sample_range_begin, num_samples);
  ThreadPool thread_pool(absl::GetFlag(FLAGS_num_reader_threads));
  ReadSamplesResult read_samples_result;
  if (!ReadSamples(sample_paths_span, &thread_pool, &read_samples_result)) {
    std::cerr << "Error: failed to read samples." << std::endl;
    return 1;
  }

  std::cout << "Read " << num_samples << " samples." << std::endl;

  const auto &samples = read_samples_result.samples;

  if (absl::GetFlag(FLAGS_run_on_cpu)) {
    for (size_t i = 0; i < num_samples - 1; ++i) {
      for (size_t j = i + 1; j < num_samples; ++j) {
        const absl::Time time_before = absl::Now();
        const float king_coeff = ComputeKing(samples[i], samples[j]);
        const absl::Time time_after = absl::Now();
        std::cout << "KING coefficient between " << i << " and " << j << ": "
                  << king_coeff << " (took " << (time_after - time_before)
                  << ")" << std::endl;
      }
    }
  } else {
    const size_t result_size = num_samples * num_samples;
    auto result = NewCudaArray<float>(result_size);
    for (size_t i = 0; i < result_size; ++i) {
      result[i] = 0.f;
    }

    const absl::Time time_before = absl::Now();

    constexpr int kCudaBlockSize = 1024;
    const int kNumCudaBlocks =
        (num_samples * num_samples + kCudaBlockSize - 1) / kCudaBlockSize;
    ComputeKingKernel<<<kNumCudaBlocks, kCudaBlockSize>>>(
        samples.get(), num_samples, result.get());

    // Wait for GPU to finish before accessing on host.
    hipDeviceSynchronize();

    const absl::Time time_after = absl::Now();

    std::vector<bool> related(num_samples);
    for (size_t i = 0; i < num_samples - 1; ++i) {
      for (size_t j = i + 1; j < num_samples; ++j) {
        // Cut off at third degree
        // (https://www.kingrelatedness.com/manual.shtml).
        constexpr float kKingCutoff = 0.0442f;
        if (result[i * num_samples + j] >= kKingCutoff) {
          related[i] = related[j] = true;
        }
      }
    }

    uint32_t num_related = 0;
    for (size_t i = 0; i < num_samples; ++i) {
      if (related[i]) {
        ++num_related;
      }
    }

    std::cout << num_related << " related samples found." << std::endl;
    std::cout << "CUDA kernel time: " << (time_after - time_before)
              << std::endl;
  }

  return 0;
}
