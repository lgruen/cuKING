#include "hip/hip_runtime.h"
#include <absl/base/thread_annotations.h>
#include <absl/flags/flag.h>
#include <absl/flags/parse.h>
#include <absl/synchronization/blocking_counter.h>
#include <absl/synchronization/mutex.h>
#include <absl/types/span.h>
#include <zstd.h>

#include <filesystem>
#include <fstream>
#include <iostream>
#include <memory>
#include <queue>
#include <string>
#include <thread>
#include <vector>

#include "cuking.h"

ABSL_FLAG(std::string, sample_list, "",
          "A text file listing one .cuking.zst input file path per line.");
ABSL_FLAG(
    size_t, sample_range_begin, 0,
    "The inclusive index of the first sample to consider in the sample list.");
ABSL_FLAG(
    size_t, sample_range_end, 0,
    "The exclusive index of the last sample to consider in the sample list.");
ABSL_FLAG(int, num_reader_threads, 100,
          "How many threads to use for parallel file reading.");

__global__ void add_kernel(int n, float *x, float *y) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

namespace {

// Adapted from the Abseil thread pool.
class ThreadPool {
 public:
  explicit ThreadPool(const int num_threads) {
    assert(num_threads > 0);
    for (int i = 0; i < num_threads; ++i) {
      threads_.push_back(std::thread(&ThreadPool::WorkLoop, this));
    }
  }

  ThreadPool(const ThreadPool &) = delete;
  ThreadPool &operator=(const ThreadPool &) = delete;

  ~ThreadPool() {
    {
      absl::MutexLock l(&mu_);
      for (size_t i = 0; i < threads_.size(); i++) {
        queue_.push(nullptr);  // Shutdown signal.
      }
    }
    for (auto &t : threads_) {
      t.join();
    }
  }

  // Schedule a function to be run on a ThreadPool thread immediately.
  void Schedule(std::function<void()> func) {
    assert(func != nullptr);
    absl::MutexLock l(&mu_);
    queue_.push(std::move(func));
  }

 private:
  bool WorkAvailable() const ABSL_EXCLUSIVE_LOCKS_REQUIRED(mu_) {
    return !queue_.empty();
  }

  void WorkLoop() {
    while (true) {
      std::function<void()> func;
      {
        absl::MutexLock l(&mu_);
        mu_.Await(absl::Condition(this, &ThreadPool::WorkAvailable));
        func = std::move(queue_.front());
        queue_.pop();
      }
      if (func == nullptr) {  // Shutdown signal.
        break;
      }
      func();
    }
  }

  absl::Mutex mu_;
  std::queue<std::function<void()>> queue_ ABSL_GUARDED_BY(mu_);
  std::vector<std::thread> threads_;
};

// RAII-style CUDA-managed array.
template <typename T>
struct CudaArray {
  explicit CudaArray(const size_t num_elements) : num_elements(num_elements) {
    const auto err = hipMallocManaged(&data, num_elements * sizeof(T));
    if (err) {
      std::cerr << "Error: can't allocate CUDA memory: "
                << hipGetErrorString(err) << std::endl;
      exit(1);
    }
  }

  CudaArray &operator=(const CudaArray &) = delete;
  CudaArray(const CudaArray &) = delete;

  CudaArray(CudaArray &&other) {
    data = other.data;
    num_elements = other.num_elements;
    other.data = nullptr;
    other.num_elements = 0;
  }

  CudaArray &operator=(CudaArray &&other) {
    if (this != &other) {
      data = other.data;
      num_elements = other.num_elements;
      other.data = nullptr;
      other.num_elements = 0;
    }
    return *this;
  }

  ~CudaArray() {
    hipFree(data);
    data = nullptr;
    num_elements = 0;
  }

  T *data = nullptr;
  size_t num_elements = 0;
};

struct Sample {
  explicit Sample(const size_t num_entries) : entries(num_entries) {}

  CudaArray<uint16_t> entries;
  uint32_t num_hets = 0;
};

// Reads and decompresses sample data from `paths`, adding entries with
// corresponding index into `result`.  Returns false if any failures occurred.
bool ReadSamples(const absl::Span<std::string> &paths,
                 ThreadPool *const thread_pool,
                 std::vector<std::unique_ptr<Sample>> *const result) {
  result->clear();
  result->resize(paths.size());

  absl::BlockingCounter blocking_counter(paths.size());
  std::atomic<bool> success(true);
  for (size_t i = 0; i < paths.size(); ++i) {
    thread_pool->Schedule([&, i] {
      const auto &path = paths[i];
      // Determine the file size.
      std::error_code error_code;
      const size_t file_size = std::filesystem::file_size(path, error_code);
      if (error_code) {
        std::cerr << "Error: failed to determine size of \"" << path
                  << "\": " << error_code << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      std::ifstream in(path, std::ifstream::binary);
      if (!in) {
        std::cerr << "Error: failed to open \"" << path << "\"." << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      // Read the entire file.
      std::vector<uint8_t> contents(file_size);
      in.read(reinterpret_cast<char *>(contents.data()), file_size);
      if (!in) {
        std::cerr << "Error: failed to read \"" << path << "\"." << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }
      in.close();

      // Validate the file header.
      cuking::FileHeader file_header;
      bool valid_header = true;
      if (contents.size() < sizeof(cuking::FileHeader)) {
        valid_header = false;
      } else {
        memcpy(&file_header, contents.data(), sizeof(cuking::FileHeader));
        for (size_t i = 0; i < sizeof(file_header.magic); ++i) {
          if (file_header.magic[i] != cuking::kExpectedMagic[i]) {
            valid_header = false;
            break;
          }
        }
      }

      if (!valid_header) {
        std::cerr << "Error: failed to validate header for \"" << path << "\"."
                  << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      // Decompress the contents.
      auto sample = std::make_unique<Sample>(file_header.decompressed_size /
                                             sizeof(uint16_t));
      sample->num_hets = file_header.num_hets;
      const size_t zstd_result =
          ZSTD_decompress(sample->entries.data, file_header.decompressed_size,
                          contents.data() + sizeof(cuking::FileHeader),
                          contents.size() - sizeof(cuking::FileHeader));
      if (ZSTD_isError(zstd_result) ||
          zstd_result != file_header.decompressed_size) {
        std::cerr << "Error: failed to decompress \"" << path << "\"."
                  << std::endl;
        success = false;
        blocking_counter.DecrementCount();
        return;
      }

      (*result)[i] = std::move(sample);
      blocking_counter.DecrementCount();
    });
  }

  blocking_counter.Wait();
  return success;
}

inline uint32_t DecodeLocusDelta(const uint16_t value) { return value >> 2; }

inline cuking::VariantCategory DecodeVariantCategory(const uint16_t value) {
  return static_cast<cuking::VariantCategory>(value & 3);
}

float ComputeKing(const Sample &sample_i, const Sample &sample_j) {
  // See https://hail.is/docs/0.2/methods/relatedness.html#hail.methods.king.
  uint32_t num_both_het = 0, num_opposing_hom = 0;
  for (uint32_t index_i = 0, index_j = 0,
                pos_i = DecodeLocusDelta(sample_i.entries.data[0]),
                pos_j = DecodeLocusDelta(sample_j.entries.data[0]);
       pos_i != static_cast<uint32_t>(-1) ||
       pos_j != static_cast<uint32_t>(-1);) {
    if (pos_i < pos_j) {
      if (DecodeVariantCategory(sample_i.entries.data[index_i]) ==
          cuking::VariantCategory::kHomAlt) {
        ++num_opposing_hom;
      }

      if (++index_i < sample_i.entries.num_elements) {
        pos_i += DecodeLocusDelta(sample_i.entries.data[index_i]);
      } else {
        pos_i = static_cast<uint32_t>(-1);
      }
    }

    if (pos_j < pos_i) {
      if (DecodeVariantCategory(sample_j.entries.data[index_j]) ==
          cuking::VariantCategory::kHomAlt) {
        ++num_opposing_hom;
      }

      if (++index_j < sample_j.entries.num_elements) {
        pos_j += DecodeLocusDelta(sample_j.entries.data[index_j]);
      } else {
        pos_j = static_cast<uint32_t>(-1);
      }
    }

    if (pos_i == pos_j && pos_i != static_cast<uint32_t>(-1)) {
      if (DecodeVariantCategory(sample_i.entries.data[index_i]) ==
              cuking::VariantCategory::kHet &&
          DecodeVariantCategory(sample_j.entries.data[index_j]) ==
              cuking::VariantCategory::kHet) {
        ++num_both_het;
      }

      if (++index_i < sample_i.entries.num_elements) {
        pos_i += DecodeLocusDelta(sample_i.entries.data[index_i]);
      } else {
        pos_i = static_cast<uint32_t>(-1);
      }

      if (++index_j < sample_j.entries.num_elements) {
        pos_j += DecodeLocusDelta(sample_j.entries.data[index_j]);
      } else {
        pos_j = static_cast<uint32_t>(-1);
      }
    }
  }

  // Return the "between-family" estimator.
  return 0.5f + (2.f * num_both_het - 4.f * num_opposing_hom -
                 sample_i.num_hets - sample_j.num_hets) /
                    (4.f * std::min(sample_i.num_hets, sample_j.num_hets));

  return 0.f;
}

}  // namespace

int main(int argc, char **argv) {
  absl::ParseCommandLine(argc, argv);

  const auto &sample_list_file = absl::GetFlag(FLAGS_sample_list);
  if (sample_list_file.empty()) {
    std::cerr << "Error: no sample list file specified." << std::endl;
    return 1;
  }

  std::ifstream sample_list(sample_list_file);
  std::string line;
  std::vector<std::string> sample_paths;
  while (std::getline(sample_list, line)) {
    if (line.empty()) {
      continue;
    }
    sample_paths.push_back(line);
  }

  const size_t sample_range_begin = absl::GetFlag(FLAGS_sample_range_begin);
  const size_t sample_range_end = absl::GetFlag(FLAGS_sample_range_end);
  if (sample_range_begin >= sample_range_end ||
      sample_range_end > sample_paths.size()) {
    std::cerr << "Error: invalid sample range specified." << std::endl;
    return 1;
  }

  ThreadPool thread_pool(absl::GetFlag(FLAGS_num_reader_threads));
  std::vector<std::unique_ptr<Sample>> samples;
  const auto sample_paths_span =
      absl::MakeSpan(sample_paths)
          .subspan(sample_range_begin, sample_range_end - sample_range_begin);
  if (!ReadSamples(sample_paths_span, &thread_pool, &samples)) {
    std::cerr << "Error: failed to read samples." << std::endl;
    return 1;
  }

  std::cout << "Read " << samples.size() << " samples." << std::endl;

  for (size_t i = 0; i < samples.size() - 1; ++i) {
    for (size_t j = i + 1; j < samples.size(); ++j) {
      std::cout << "KING coefficient between " << i << " and " << j << ": "
                << ComputeKing(*samples[i], *samples[j]) << std::endl;
    }
  }

  constexpr int N = 1 << 20;

  // Allocate Unified Memory – accessible from CPU or GPU.
  CudaArray<float> x(N), y(N);

  // Initialize x and y arrays on the host.
  for (int i = 0; i < N; i++) {
    x.data[i] = 1.0f;
    y.data[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU.
  constexpr int blockSize = 256;
  constexpr int numBlocks = (N + blockSize - 1) / blockSize;
  add_kernel<<<numBlocks, blockSize>>>(N, x.data, y.data);

  // Wait for GPU to finish before accessing on host.
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f).
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = std::fmax(maxError, std::fabs(y.data[i] - 3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  return 0;
}
